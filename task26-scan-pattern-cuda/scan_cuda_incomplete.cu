
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
TEMPO SEQUENCIAL:

real    0m0.414s
user    0m0.185s
sys     0m0.221s

TEMPO CUDA:

real    0m2.330s
user    0m1.143s
sys     0m1.099s

*/

__global__ void scan_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  double x;

  // cria vetor na memória local
  __shared__ double p[1024];

  // carrega elementos do vetor da memória global para a local
  if(b+t < width)
    p[t] = a[b+t];

  // espera que todas as threads tenham carregado seus elementos
  __syncthreads();

  for (int i = 1; i < blockDim.x; i *= 2) { // realiza o scan em log n passos
    if(t >= i) // verifica se a thread ainda participa neste passo
      x = p[t] + p[t-i]; // atribui a soma para uma variável temporária

    __syncthreads(); // espera threads fazerem as somas

    if(t >= i)
      p[t] = x; // copia a soma em definitivo para o vetor local

    __syncthreads();
  }

  if(b + t < width) // copia da memória local para a global
    a[b+t] = p[t];

  if(t == blockDim.x-1) // se for a última thread do bloco
    s[blockIdx.x+1] = a[b+t]; // copia o seu valor para o vetor de saída
} 

__global__ void add_cuda(double *a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  
  // soma o somatório do último elemento do bloco anterior ao elemento atual
  if(b+t < width)
    a[b+t] += s[blockIdx.x];
}

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocar vetores "a" e "s" no device
  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_s, s_size);

  // copiar vetor "a" para o device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  // definição do número de blocos e threads (dimGrid e dimBlock)
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // chamada do kernel scan
  scan_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar vetor "s" para o host
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);

  // scan no host (já implementado)
  s[0] = 0;
  for (int i = 1; i < num_blocks; i++)
    s[i] += s[i-1];
 
  // copiar vetor "s" para o device
  hipMemcpy(d_s, s, s_size, hipMemcpyHostToDevice);

  // chamada do kernel da soma
  add_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);

  // copiar o vetor "a" para o host
  hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

  printf("\na[%d] = %f\n",width-1,a[width-1]);
  
  hipFree(d_a);
  hipFree(d_s);
}
